#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <assert.h>
#include <omp.h>
#include <sys/time.h>

__global__ void
matrixMulCUDAkernel(float *C, float *A, float *B, int n)
{
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for ( int e = 0; e < n; ++e)
        Cvalue += A[row * n + e] * B[e * n + col];
    C[row * n + col] = Cvalue;
}

void matrixMulCPU_serial(float *C, float *A, float *B, int n)
{
    for (int i = 0; i < (int)(n*n); i++)
    {
	
        float temp = 0;
        int x = i % n;
        int y = i / n;
        for( int k = 0; k < (int)(n); k++)
        {
            temp += A[n * y + k] * B[n * k + x];
        }

        C[n*y + x] = temp;
    }
}

void matrixMulCPU_parallel(float *C, float *A, float *B, int n)
{
    omp_set_num_threads(16);
    #pragma omp parallel for
    for (int i = 0; i < (int)(n*n); i++)
    {
	
        float temp = 0;
        int x = i % n;
        int y = i / n;
        for( int k = 0; k < (int)(n); k++)
        {
            temp += A[n * y + k] * B[n * k + x];
        }

        C[n*y + x] = temp;
    }
}

void matrixMulCUDA( float *h_C, float *h_A, float *h_B, int block_size, int n)
{
    unsigned int mem_size = sizeof(float) * n * n;
    // Allocate device memory
    float *d_A, *d_B, *d_C;

    hipMalloc((void **) &d_A, mem_size);
    hipMalloc((void **) &d_B, mem_size);
    hipMalloc((void **) &d_C, mem_size);

    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size, hipMemcpyHostToDevice);

    // Setup execution parameters
    dim3 threads(block_size, block_size); // (block_size, block_size, 1) default setting 1
    dim3 grid(n / threads.x, n / threads.y);

    // Execute the kernel
    matrixMulCUDAkernel<<< grid, threads >>>(d_C, d_A, d_B, n);

    hipDeviceSynchronize();

    // Copy result from device to host
    hipMemcpy(h_C, d_C, mem_size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

__global__ void
matrixMulTiledCUDAkernel(float *C, float *A, float *B, int tile_size, int n) 
{
    // Shared memory for tiles
    __shared__ float ds_A[tile_size][tile_size];
    __shared__ float ds_B[tile_size][tile_size];

    int row = blockIdx.y * tile_size + threadIdx.y;
    int col = blockIdx.x * tile_size + threadIdx.x;

    float Cvalue = 0;

    // Need to proceed loading data for tile n/tilie_size times
    for (int e = 0; e < n/tile_size; ++e) {
        // Load from A & B to tile
        ds_A[threadIdx.y][threadIdx.x] = A[row*n + (e * tile_size) + threadIdx.x];
        ds_B[threadIdx.y][threadIdx.x] = B[(e * tile_size + threadIdx.y) * n + col];
        __syncthreads();

        // Partial computation
        for (int i = 0; i < tile_size; ++i) {
            Cvalue += ds_A[threadIdx.y][i] * ds_B[i][threadIdx.x];
        }
        __syncthreads();
    }
    C[row * n + col] = Cvalue;

}

void matrixMulCUDA_tiled( float *h_C, float *h_A, float *h_B, int block_size, int n)
{
    // TODO: implement the code for tiled matrix multiplication

    unsigned int mem_size = sizeof(float) * n * n;
    // Allocate device memory
    float *d_A, *d_B, *d_C;

    hipMalloc((void **) &d_A, mem_size);
    hipMalloc((void **) &d_B, mem_size);
    hipMalloc((void **) &d_C, mem_size);

    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size, hipMemcpyHostToDevice);

    // Setup execution parameters
    dim3 threads(block_size, block_size); // (block_size, block_size, 1) default setting 1
    dim3 grid(n / threads.x, n / threads.y);

    matrixMulTiledCUDAkernel<<<grid, threads>>>(d_C, d_A, d_B, block_size, n)

    hipDeviceSynchronize();

    // Copy result from device to host
    hipMemcpy(h_C, d_C, mem_size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

bool error_checking(float *reference, float *result, int n, double eps)
{
    printf("Checking for error with eps=%.1e\n",eps);

    bool correct = true;
    for (int i = 0; i < (int)(n * n); i++)
    {
        double abs_err = fabs(reference[i] - result[i]);
        double abs_val = fabs(result[i]);
        double rel_err = abs_err/abs_val;

        if(rel_err > eps){
            // Remove comment if you want detailed error report
            //int x = i % n;
            //int y = i / n;
            //printf("Error! Matrix[%d][%d]=%.8f, ref=%.8f\n", y,x, result[i], reference[i]);
	    correct = false;
        }
    }

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    return correct;
}

void initMatrix(float *data, int size)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = ((float)random())/10000;
    }
}

void initMatrix0(float *data, int size)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = 0;
    }
}

float get_elapsed_time(struct timeval start_time, struct timeval stop_time)
{
    float elapsed_time = (stop_time.tv_sec - start_time.tv_sec) +
                (float)(stop_time.tv_usec - start_time.tv_usec)/1000000;

    return elapsed_time;
}

int matrixMultiply(int block_size, int n)
{
    // Allocate memory for matrices A and B
    unsigned int size = n * n;
    unsigned int mem_size = sizeof(float) * size;
    float *matrix_A = (float *)malloc(mem_size);
    float *matrix_B = (float *)malloc(mem_size);

    bool correctness = true;

    // Initialize matrix A, B
    srandom(0);
    initMatrix(matrix_A, size);
    initMatrix(matrix_B, size);

    // Allocate matrix C
    float *matrix_C = (float *) malloc(mem_size);

    float *reference = (float *) malloc(mem_size);
    initMatrix0(reference, size);

    struct timeval start_time;
    struct timeval stop_time;
    
    
    printf("\nStarting CPU serial\n");
    gettimeofday(&start_time, NULL);

    matrixMulCPU_serial(reference, matrix_A, matrix_B, n);

    gettimeofday(&stop_time, NULL);
    printf("Ended CPU serial\n");

    float elapsed_time = get_elapsed_time(start_time, stop_time);
    printf("CPU serial execution time:%f\n",elapsed_time);

    printf("Use CPU serial result as error checking reference\n");



    initMatrix0(matrix_C, size);

    printf("\nStarting CPU parallel\n");
    gettimeofday(&start_time, NULL);

    matrixMulCPU_parallel(matrix_C, matrix_A, matrix_B, n);

    gettimeofday(&stop_time, NULL);
    printf("Ended CPU parallel\n");

    elapsed_time = get_elapsed_time(start_time, stop_time);
    printf("CPU parallel execution time:%f\n", elapsed_time);

    correctness &= error_checking(reference, matrix_C, n, 0);



    initMatrix0(matrix_C, size);

    printf("\nStarting GPU baseline\n");
    gettimeofday(&start_time, NULL);

    matrixMulCUDA(matrix_C, matrix_A, matrix_B, block_size, n);    

    gettimeofday(&stop_time, NULL);
    printf("Ended GPU baseline\n");

    elapsed_time = get_elapsed_time(start_time, stop_time);
    printf("GPU baseline execution time:%f\n", elapsed_time);

    correctness &= error_checking(reference, matrix_C, n, 1.e-6);



    initMatrix0(matrix_C, size);

    printf("\nStarting GPU tiled\n");
    gettimeofday(&start_time, NULL);

    matrixMulCUDA_tiled(matrix_C, matrix_A, matrix_B, block_size, n);

    gettimeofday(&stop_time, NULL);
    printf("Ended GPU tiled\n");

    elapsed_time = get_elapsed_time(start_time, stop_time);
    printf("GPU tiled execution time:%f\n", elapsed_time);

    correctness &= error_checking(reference, matrix_C, n, 1.e-6);


  // Clean up memory
    free(matrix_A);
    free(matrix_B);
    free(matrix_C);
    free(reference);

    hipDeviceReset();

    if(correctness==true)
        return EXIT_SUCCESS;
    else
        return EXIT_FAILURE;
}


/**
 * Program main
 */
int main(int argc, char **argv)
{
    printf("[Matrix Multiply] - Starting...\n");

    if (argc != 2) {
        printf("Usage: %s <num_row/column> (must be multiple of 32)",argv[0]);
        exit(EXIT_FAILURE);
    }
    int n = atoi(argv[1]);
    if (n < 32 || n%32 != 0){
        printf("<num_row/column> must be multiple of 32");
        exit(EXIT_FAILURE);
    }

    int block_size = 32;

    printf("Matrix(%d,%d)\n", n, n);

    int matrix_result = matrixMultiply(block_size, n);

    exit(matrix_result);
}
